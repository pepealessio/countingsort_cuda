#include "hip/hip_runtime.h"
// Course:           High Performance Computing
// A.Y:              2021/22
// Lecturer:         Francesco Moscato           fmoscato@unisa.it

// Team:
// Alessio Pepe          0622701463      a.pepe108@studenti.unisa.it
// Teresa Tortorella     0622701507      t.tortorella3@studenti.unisa.it
// Paolo Mansi           0622701542      p.mansi5@studenti.unisa.it

// Copyright (C) 2021 - All Rights Reserved

// This file is part of Counting_Sort.

// Counting_Sort is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.

// Counting_Sort is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.

// You should have received a copy of the GNU General Public License
// along with Counting_Sort.  If not, see <http://www.gnu.org/licenses/>.

/**
 * @file    counting_sort.c
 * @author  Alessio Pepe         (a.pepe108@studenti.unisa.it)
 * @author  Paolo Mansi          (p.mansi5@studenti.unisa.it)
 * @author  Teresa Tortorella    (t.tortorella3@studenti.unisa.it)
 * @version 1.0.0
 * @date 2022-01-24
 * 
 * @copyright Copyright (c) 2022
 * 
 */


#include <stdio.h>
#include <assert.h>
#include <unistd.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <sys/time.h>

/**
 * @brief Use start_time with an non used id to start measure time in that point of the code.
 * 
 */
#define STARTTIME(id)                             \
   struct timeval start_time_##id, end_time_##id; \
   gettimeofday(&start_time_##id, NULL);

/**
 * @brief Use end_tipe with a previous used id to stop measure time in that point of the code.
 *        The value of time will be saved in x.
 * 
 */
#define ENDTIME(id, x)                 \
   gettimeofday(&end_time_##id, NULL); \
   x = ((end_time_##id.tv_sec  - start_time_##id.tv_sec) * 1000000u +  end_time_##id.tv_usec - start_time_##id.tv_usec) / 1.e6;


#define FIXED_ARRAY

#define MIN(a,b) (((a)<(b))?(a):(b))
#define MAX(a,b) (((a)>(b))?(a):(b))


texture <int, 1> pmfTextRef;


void cudaGetError()
{
    hipError_t err = hipGetLastError();  
    if (err != hipSuccess) 
    {
        printf("CUDA error: %s\n", hipGetErrorString(err)); 
    }
}


#define cudaCheck(status, err)                              \
    if (status != hipSuccess) {                            \
        fprintf(stderr, "CUDA check failed: %s\n", err);    \
        cudaGetError();                                     \
        exit(1);                                            \
    }


#define cudaStartTime(id)               \
    hipEvent_t start##id, stop##id;     \
    hipEventCreate(&start##id);        \
    hipEventCreate(&stop##id);         \
    hipEventRecord(start##id);         \


#define cudaStopTime(id)                \
    hipEventRecord(stop##id);          \
    hipEventSynchronize(stop##id);     \


#define cudaElapsedTime(id, x)                              \
    hipEventElapsedTime(&x, start##id, stop##id);          \
    hipEventDestroy(start##id);                            \
    hipEventDestroy(stop##id);                             \


void init_rand_vector(int *A, int A_len, int min_value, int max_value)
{ 
    #ifdef FIXED_ARRAY
    srand(1256765);
    #endif
 
    for (unsigned int i = 0; i < A_len; i++)
    {
        A[i] = min_value + (rand() % (max_value - min_value + 1));
    }
}

void printV(int *array, int len)
{
    for (unsigned int i = 0; i < len; i++)
    {
        printf("%d ", array[i]);
    }
    printf("\n");
}


__global__ void max_min(int *d_max_A, int *d_min_A, int *d_data, int d_data_len)
{
    extern __shared__ int arr[];
    int *s_min = arr;
    int *s_max = arr + blockDim.x;

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    // each thread loads one element from global to shared mem
    s_min[tid] = d_data[i];
    s_max[tid] = d_data[i];
    __syncthreads();

    // do reduction in shared mem
    for (unsigned int s = blockDim.x/2; s > 0; s >>= 1) 
    {  
        if ((tid < s) && ((i + s) < d_data_len))
        {
            s_min[tid] = MIN(s_min[tid], s_min[tid + s]);
            s_max[tid] = MAX(s_max[tid], s_max[tid + s]);
        }
        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0)
    {
        d_min_A[blockIdx.x] = s_min[0];
        d_max_A[blockIdx.x] = s_max[0];
    }
}

__global__ void max_min_red(int *d_max_A, int *d_min_A, int d_len)
{
    extern __shared__ int arr[];
    int *s_min = arr;
    int *s_max = arr + blockDim.x;

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    // each thread loads one element from global to shared mem
    s_min[tid] = d_min_A[i];
    s_max[tid] = d_max_A[i];
    __syncthreads();

    // do reduction in shared mem
    for (unsigned int s = blockDim.x/2; s > 0; s >>= 1) 
    {  
        if ((tid < s) && ((i + s) < d_len))
        {
            s_min[tid] = MIN(s_min[tid], s_min[tid + s]);
            s_max[tid] = MAX(s_max[tid], s_max[tid + s]);
        }
        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0)
    {
        d_min_A[blockIdx.x] = s_min[0];
        d_max_A[blockIdx.x] = s_max[0];
    } 
}


__global__ void pmf_count(int *d_data, int d_data_len, int *d_data_max, int *d_data_min, int *d_pmf_data)
{
    // init a shared pmf array for each block
    extern __shared__ int s_pmf[];

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    // set initial value to 0
    int range = *d_data_max - *d_data_min + 1;

    for (int offset = 0; offset < range; offset += blockDim.x)
    {
        if ((tid + offset) < range)
        {
            s_pmf[tid + offset] = 0;
        }
    }

    // wait until all threads have completed the initialization process
    __syncthreads();

    // increment the local pdf array
    if (i < d_data_len)
    {
        atomicAdd(&s_pmf[d_data[i] - *d_data_min], 1);
    }
    
    // wait until all threads have completed the counting process
    __syncthreads();

    // merge the various pdf array
    for (int offset = 0; offset < range; offset += blockDim.x)
    {
        if ((tid + offset) < range)
        {
            atomicAdd(&d_pmf_data[tid + offset], s_pmf[tid + offset]);
        }
    }
}


__global__ void scan(int *d_pmf, int d_len)
{
    extern __shared__ int scan_a[];

    int i, j, tid;

    tid = threadIdx.x;
    j = blockIdx.x * (2 * blockDim.x) + threadIdx.x;

    // Copy array in block
    if (j < d_len)
    {
        scan_a[tid] = d_pmf[j];
    }
    
    if ((j + blockDim.x) < d_len)
    {
        scan_a[tid + blockDim.x] = d_pmf[j + blockDim.x];
    }

    __syncthreads();

    // Scan 
    for (int stride = 1; stride <= blockDim.x; stride <<= 1)
    {
        i = (threadIdx.x + 1) * stride * 2 - 1;
        if (i < 2 * blockDim.x)
        {
            scan_a[i] += scan_a[i - stride];
        }
        __syncthreads();
    }

    // Post scan
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        i = (threadIdx.x + 1) * stride * 2 - 1;
        if ((i + stride) < 2 * blockDim.x)
        {
            scan_a[i + stride] += scan_a[i];
        }
        __syncthreads();
    }

    // Copy partially cdf in the global memory
    if (j < d_len)
    {
        d_pmf[j] = scan_a[tid];
    }

    if ((j + blockDim.x) < d_len)
    {
        d_pmf[j + blockDim.x] = scan_a[tid + blockDim.x];
    }
}

__global__ void scan_red(int *d_pmf, int d_len, int stride)
{
    // First block was already complete
    if (blockIdx.x % 2 == 0)
    {
        return;
    }

    // Copy last element of the previous block on all block element.
    int i = blockIdx.x * (stride * blockDim.x) + threadIdx.x;
    int prec_sum = d_pmf[blockIdx.x * (stride * blockDim.x) - 1];

    for (int j = 0; j < stride; j++)
    {
        if ((i + j * blockDim.x) < d_len)
        {
            d_pmf[i + j * blockDim.x] += prec_sum;
        }
    }
}

__global__ void populate(int *d_data, int *d_data_min, int *d_cdf, int d_cdf_len)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < d_cdf_len)
    {
        int start = i != 0 ? d_cdf[i-1] : 0;
        for (int j = 0; j < d_cdf[i] - start; j++)
        {
            d_data[start + j] = *d_data_min + i;
        }
    }
}

__global__ void populate_text(int *d_data, int *d_data_min, int d_cdf_len)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < d_cdf_len)
    {
        int start = i != 0 ? tex1Dfetch(pmfTextRef, i-1) : 0;
        for (int j = 0; j < tex1Dfetch(pmfTextRef, i) - start; j++)
        {
            d_data[start + j] = *d_data_min + i;
        }
    }
}


/**
 * This GPU kernel takes an array of states, and an array of ints, and puts a random int into each 
 */
__global__ void randoms(int* numbers, int len, int min, int max, int seed) 
{   
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    // hiprand works like rand - except that it takes a state as a parameter
    //hiprandState state;
    if (i < len)
    {
        //hiprand_init(seed, i, 0, &state);
        if (i % 3 == 0)
        {
            numbers[i] = min + (i * seed / blockDim.x) % (max - min + 1);
        }
        else if (i % 3 == 1)
        {
            numbers[i] = min + (i + seed / blockDim.x) % (max - min + 1);
        }
        else
        {
            numbers[i] = min + (seed / blockDim.x - i) % (max - min + 1);
        }
    }  
}

void cuda_init_rand_vector(int gridSize, int blockSize, int *h_A, int h_len, int min, int max)
{
    /* allocate an array of unsigned ints on the CPU and GPU */
    int *d_A1;
    hipMalloc((void**) &d_A1, h_len * sizeof(int));

    /* invoke the kernel to get some random numbers */
    randoms <<<gridSize, blockSize>>> (d_A1, h_len, min, max, 2342);

    /* copy the random numbers back */
    hipMemcpy(h_A, d_A1, h_len * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_A1);
}


// main
int main(int argc, char** argv) {

    // ------------------------------------------------------
    // Read parameter from argv
    if (argc < 8)
    {
        printf("USAGE: %s len min max blockMinMAx blockPmf blockScan blockPopulate\n", argv[0]);
        exit(1);
    }
    int min = atoi(argv[2]);  // Just for generation 
    int max = atoi(argv[3]);  // Just for generation
    int h_len = atoi(argv[1]);
    // Bench parameters
    int blockMinMax = atoi(argv[4]);
    int blockPmf = atoi(argv[5]);
    int blockScan = atoi(argv[6]);
    int blockPopulate = atoi(argv[7]);

    // --------------- Random Array Generation --------------
    int *h_A;
    h_A = (int *) malloc(h_len * sizeof(int));
    //init_rand_vector(h_A, h_len, min, max); // sequential init
    cuda_init_rand_vector((int) ceilf((float) h_len / (float) blockMinMax), blockMinMax, h_A, h_len, min, max);
    cudaGetError();
    //printV(h_A, h_len);  // Debug print

    double t_algo;
    STARTTIME(0);

    cudaStartTime(0);

    // -------------- Define block and grid size ------------
    // The block size was the maximum (1024). Grid size was
    // selected dinamically to cover all tha array.
    dim3 blockSizeMinMax(blockMinMax);
    dim3 gridSizeMinMax((int) ceilf((float) h_len / (float) blockSizeMinMax.x));

    // ------------ Allocate space on cuda ------------------
    // - Array
    // - Array of local max and min
    // - global max and min
    int *d_A, *d_min_A, *d_max_A, *d_min, *d_max;

    cudaCheck( hipMalloc((void **)&d_A, h_len * sizeof(int)), "Allocation d_A" );
    cudaCheck( hipMalloc((void **)&d_min_A, gridSizeMinMax.x * sizeof(int)), "Allocation d_min_A" );
    cudaCheck( hipMalloc((void **)&d_max_A, gridSizeMinMax.x * sizeof(int)), "Allocation d_max_A" );
    cudaCheck( hipMalloc((void **)&d_min, sizeof(int)), "Allocation d_min" );
    cudaCheck( hipMalloc((void **)&d_max, sizeof(int)), "Allocation d_max" );

    // ------------- Copy array to gpu ----------------------
    cudaCheck( hipMemcpy(d_A, h_A, h_len * sizeof(int), hipMemcpyHostToDevice), "memcpy h_A to d_A");

    // ------------- Max&Min Kernels ------------------------
    cudaStartTime(1);

    max_min <<<gridSizeMinMax, blockSizeMinMax, 2 * blockSizeMinMax.x * sizeof(int) >>> (d_max_A, d_min_A, d_A, h_len); // Now we have an array of gridSize.x local minimum
    cudaGetError();

    int old_grid_size;
    dim3 redGridSize(gridSizeMinMax.x);
    do 
    {
        old_grid_size = redGridSize.x;
        redGridSize.x = (int) ceilf((float) redGridSize.x / (float) gridSizeMinMax.x);
        // printf("Running with %d, %d, %d\n", redGridSize.x, blockSizeMinMax.x, old_grid_size);
        max_min_red <<<redGridSize, blockSizeMinMax, 2 * blockSizeMinMax.x * sizeof(int)>>> (d_max_A, d_min_A, old_grid_size);
        cudaGetError();
    }
    while (redGridSize.x != 1);

    cudaStopTime(1);

    // ----------- DEBUG: Print max and min ---------------------
    /*int *h_max_A, *h_min_A;

    h_max_A = (int *) malloc( sizeof(int));
    h_min_A = (int *) malloc( sizeof(int));

    cudaCheck(hipMemcpy((void *) h_min_A, (const void *)d_min_A, sizeof(int), hipMemcpyDeviceToHost), "memcpy h_min_A");
    cudaCheck(hipMemcpy((void *) h_max_A, (const void *)d_max_A, sizeof(int), hipMemcpyDeviceToHost), "memcpy h_max_A");

    printf("Min: ");
    printV(h_min_A, 1); //gridSize.x);
    printf("\nMax: ");
    printV(h_max_A, 1); //gridSize.x);
    printf("\n");

    free(h_max_A);
    free(h_min_A);
    // ----------------------------------------------------------*/

    // -------------- Compute PMF --------------------------------
    int h_max, h_min;
    cudaCheck( hipMemcpy((void *) &h_max, (const void *)d_max_A, sizeof(int), hipMemcpyDeviceToHost), "memcpy d_max_A[0] to h_max");
    cudaCheck( hipMemcpy((void *) &h_min, (const void *)d_min_A, sizeof(int), hipMemcpyDeviceToHost), "memcpy d_min_A[0] to h_min");
    
    int range_size = (h_max - h_min + 1);

    int *d_pmf;
    cudaCheck( hipMalloc((void **) &d_pmf, range_size * sizeof(int)), "Allocate d_pmf" );
    cudaCheck( hipMemset((void *) d_pmf, 0, range_size * sizeof(int)), "memcpy d_pmf" );  

    dim3 blockSizePmf(blockPmf);
    dim3 gridSizePmf((int) ceilf((float) h_len / (float) blockSizePmf.x));

    cudaStartTime(2);
    pmf_count <<< gridSizePmf, blockSizePmf, range_size * sizeof(int) >>> (d_A, h_len, d_max_A, d_min_A, d_pmf);
    cudaGetError();
    cudaStopTime(2);

    // ---------------- CDF calculate ------------------------
    dim3 cdfBlockDim(blockScan);
    dim3 cdfGridDim((int) ceilf((float) range_size / (float) cdfBlockDim.x * 2.f));   

    cudaStartTime(3);
    scan <<< cdfGridDim, cdfBlockDim, 2 * cdfBlockDim.x * sizeof(int) >>> (d_pmf, range_size);
    cudaGetError();

    int new_gridDim = cdfGridDim.x;
    int stride = 2;
    while (new_gridDim != 1)
    {
        cdfGridDim.x = new_gridDim % 2 == 0 ? new_gridDim : new_gridDim - 1;

        scan_red <<< cdfGridDim, cdfBlockDim >>> (d_pmf, range_size, stride);
        cudaGetError();

        new_gridDim = (int) ceilf((float) new_gridDim / 2.f);
        stride *= 2;
    }
    cudaStopTime(3);

    // Debug print 
    /*int *h_pmf;
    h_pmf = (int *) malloc(range_size * sizeof(int));
    cudaCheck(hipMemcpy(h_pmf, d_pmf, range_size * sizeof(int), hipMemcpyDeviceToHost), "memcpy d_pmf to h_pmf");
    int k = 0;
    while (k < range_size)
    {
        printV(h_pmf+k, 1024);
        k+= 1024;
        if (k%2048 == 0) printf("\n");
    } 
    free(h_pmf);    // */

    // --------------- Populate array ------------------------
    // To populate we use d_pdf in texture memory
    hipChannelFormatDesc pmfChRef = hipCreateChannelDesc <int> ();
    cudaCheck( hipBindTexture(0, pmfTextRef, d_pmf, pmfChRef), "bindTexture d_pmf" );

    dim3 populateBlockSize(blockPopulate);
    dim3 populateGridSize((int) ceilf((float) range_size / (float) populateBlockSize.x));

    cudaStartTime(4);
    populate_text <<<populateGridSize, populateBlockSize>>> (d_A, d_min_A, /*d_pmf,*/ range_size);
    cudaGetError();
    cudaStopTime(4);

    cudaCheck( hipUnbindTexture(pmfTextRef), "unbind texture d_pmf" );

    // --------------- Copy array to CPU ---------------------
    cudaCheck( hipMemcpy(h_A, d_A, h_len * sizeof(int), hipMemcpyDeviceToHost), "memcpy d_A to h_A");   

    // ------------------- Free --------------------------
    cudaCheck( hipFree(d_A), "Free d_A" );
    cudaCheck( hipFree(d_min_A), "Free d_min_A" );
    cudaCheck( hipFree(d_max_A), "Free d_max_A" );
    cudaCheck( hipFree(d_min), "Free d_min" );
    cudaCheck( hipFree(d_max), "Free d_max" );
    cudaCheck( hipFree(d_pmf), "Free d_pmf_A" );  

    cudaStopTime(0);
    
    ENDTIME(0, t_algo);

    // ------------------- Test working properly ------------
    int flag = 1;
    for (unsigned int i = 1; flag && i < h_len; i++)
    {
        if (h_A[i-1] > h_A[i])
        {
            printf("0");
        }
    }

    float t0, t1, t2, t3, t4;
    cudaElapsedTime(0, t0);
    cudaElapsedTime(1, t1);
    cudaElapsedTime(2, t2);
    cudaElapsedTime(3, t3);
    cudaElapsedTime(4, t4);

    printf("%d,%d,%d,%d,%d,%d,%d,%f,%f,%f,%f,%f,%f\n", h_len, range_size, blockMinMax, blockPmf, blockScan, blockPopulate, flag, t_algo, t0, t1, t2, t3, t4);

    free(h_A);

    return 0;
}






























